﻿
#include "hip/hip_runtime.h"


#include <iostream>
#include <fstream>
#include <vector>

#include <cassert>

// Use (void) to silence unused warnings.
#define assertm(exp, msg) assert(((void)msg, exp))

// define the size of the image
// assume square
#define ROWS 1024
#define COLS 1024

// constant for GPU mem
#define FILTER_RADIUS 2
__constant__ float FILTER[2 * FILTER_RADIUS + 1][2 * FILTER_RADIUS + 1];


__global__ void blurImage(int* input, int* output, float* filter, int radius, size_t width, size_t height) {
  // by using a global index to the total size of the array
  // we get the position by using the threadIDx, blockIdx* block Dimx 
  // same goes for y index
  int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
  int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
  float res = 0;
    for (int filterRow = 0; filterRow < 2*radius+1; filterRow++) {
      for (int filterCol = 0; filterCol < 2 * radius + 1; filterCol++) {
        int inputRow = y_idx - radius + filterRow;
        int inputCol = x_idx - radius + filterCol;
        if (inputRow >= 0 && inputRow < height && inputCol >=0 && inputCol < width){
          res += filter[(filterRow * 2 * radius + 1) + filterCol] * input[inputRow * width + inputCol];
      }
    }
  }
  output[y_idx*width+x_idx] = (int)res;
}

/* PGM format spec:
  * https://users.wpi.edu/~cfurlong/me-593n/pgmimage.html
  * header starts with P5 or P2 (ASCII)
  * width height
  * max_val
  * raw_data
  */
bool writePGM(const std::string& filename, int* data, size_t rows, size_t cols) {
  std::ofstream file(filename, std::ios::out);
  if (!file.is_open()) {
    std::cerr << "Failed to open file: " << filename << std::endl;
    return false;
  }

  // PGM header
  // ASCII
  file << "P2" << std::endl;
  file << "# " << "test.pgm" << std::endl;
  file << cols << " " << rows << std::endl;
  // 8 bit
  file << 255 << std::endl;

  // Write pixel values
  for (size_t x = 0; x < rows; x++) {
    for (size_t y = 0; y < cols; y++) {
      file << data[x * cols + y] << " ";
    }
    file << std::endl;
  }

  file.close();
  return true;
}

void drawDiag(int* data, int rows, int cols, int width) {
  for (int i = 0; i < rows; ++i) {
    // Calculate the range of columns to modify for the current row
    int start_col = std::max(0, i - width);
    int end_col = std::min(cols - 1, i + width);

    for (int j = start_col; j <= end_col; ++j) {
      // Set the value at the corresponding 1D array index
      data[i * cols + j] = 0;
    }
  }
}

int main() {
  // creates white image
  int* data = new int[ROWS * COLS]{ 255 };

  drawDiag(data, ROWS, COLS, 2);
  bool img_file_created = writePGM("test.pgm", data, ROWS, COLS);
  assertm(img_file_created == true, "Image file created\n!");

  int* h_input = new int[ROWS * COLS];
  int* h_output = new int[ROWS * COLS];

  int filter_byte_size = sizeof(float) * (2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1);
  int input_byte_size = sizeof(int) * ROWS * COLS;

  for (int i = 0; i < ROWS; i++) {
    for (int j = 0; j < COLS; j++) {
      h_input[i * COLS + j] = data[i * COLS + j];
    }
  }

  // initialize normalizecd filter 
  float* h_filter = new float[(2 * FILTER_RADIUS + 1)*(2 * FILTER_RADIUS + 1)];
  
  for (size_t i = 0; i < (2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1); i++) {
    h_filter[i] = static_cast<float>(1.0f / 9.0f);
  }


  // GPU arrs
  int* d_input;
  int* d_output;

  // gpu alloc
  hipMalloc(&d_input, input_byte_size);
  hipMalloc(&d_output, input_byte_size);
  // inform data will NOT be changed
  hipMemcpyToSymbol(HIP_SYMBOL(FILTER), h_filter, filter_byte_size);

  // Copy data from the host to the device
  hipMemcpy(d_input, h_input, input_byte_size, hipMemcpyHostToDevice);
  // might not need this
  //cudaMemcpy(FILTER, h_filter, filter_byte_size, cudaMemcpyHostToDevice);

  // Define block and grid dimensions
  dim3 blockDim(16, 16);  // 16x16 threads per block
  dim3 gridDim((COLS + blockDim.x - 1) / blockDim.x, (ROWS + blockDim.y - 1) / blockDim.y);

  // Launch the kernel
  blurImage <<<gridDim, blockDim>>>(d_input, d_output, h_filter, FILTER_RADIUS, COLS, ROWS);

  // Copy the result back to the host
  hipMemcpy(h_output, d_output, input_byte_size, hipMemcpyDeviceToHost);

  bool blurred_img = writePGM("blur.pgm", h_output, ROWS, COLS);
  assertm(blurred_img == true, "Image file is blurred!\n");

  // Free device memory
  hipFree(d_input);
  hipFree(d_output);

  delete[] h_input;
  delete[] h_output;
  delete[] h_filter;
  return 0;
}