#include "hip/hip_runtime.h"
﻿
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

// define the size of the image
#define ROWS 1024
#define COLS 1024

// define threads per block 
#define HILOS_POR_BLOQUE 512

__global__ void blurImage(float* input, float* output, size_t width, size_t height) {
  // by using a global index to the total size of the array
  // we get the position by using the threadIDx, blockIdx* block Dimx 
  // same goes for y index
  size_t x_idx = threadIdx.x + blockIdx.x * blockDim.x;

  // maybe dim 2 index? size_t y_idx = threadIdx.y + blockIdx.y * blockDim.y;
  // the number of threads might not match so we need to guard that limit
  if (x_idx < width) {
    float sum = 0;
    size_t count = 0;
  }
}

bool make_img() {
  /* PGM format spec:
  * https://users.wpi.edu/~cfurlong/me-593n/pgmimage.html
  * header starts with P5
  * width height
  * max_val
  * raw_data
  */
  const char* filename = "black_diagonal.pgm";
  /* 2D img array, no color */
  unsigned char data[COLS][ROWS];

  // bitshift value format
  const int bit_format = 8;
  const int max_val = (1 << bit_format) - 1;

  FILE* fp;
  const char* comment = "#Diagonal black line";

  /* black diagonal */
  for (size_t y = 0; y < COLS; ++y) {
    for (size_t x = 0; x < ROWS; ++x) {
      if (x == y) {
        data[y][x] = 0; // black
      }
      else {
        data[y][x] = max_val; // white
      }
    }
  }

  /* write the whole data array to ppm file in one step */
  /* create new file, give it a name and open it in binary mode */
  fp = fopen(filename, "wb");
  /* write header to the file */
  fprintf(fp, "P5\n %s\n %d\n %d\n %d\n", comment, COLS, ROWS,
    max_val);
  /* write image data bytes to the file */
  fwrite(data, sizeof(data), 1, fp);
  fclose(fp);

  return true;
}


int main() {
  bool img_file_created = make_img();
  if (img_file_created) {
    int* h_a, * h_b, * h_c;
    int* d_a, * d_b, * d_c;

    //size_t arr_size = ROWS * sizeof(float);

    hipMalloc((void**)&d_a, arr_size);
    hipMalloc((void**)&d_b, arr_size);
    hipMalloc((void**)&d_c, arr_size);

    h_a = (int*)malloc(arr_size);
    h_b = (int*)malloc(arr_size);
    h_c = (int*)malloc(arr_size);


    hipMemcpy(d_a, h_a, arr_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arr_size, hipMemcpyHostToDevice);


    //blurImage << < N / HILOS_POR_BLOQUE, HILOS_POR_BLOQUE >> > (d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, arr_size, hipMemcpyDeviceToHost);


    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
  }
  else {
    printf("Could not create file");
    return 0;
  }
}